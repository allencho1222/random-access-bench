#include "hip/hip_runtime.h"
//
// Author     :  matto@xilinx 14JAN2018, alai@xilinx 25JULY2018
// Filename   :  indirectTest_onlyGPU.cu
// Description:  Cuda random access benchmark example based on indirect.c by gswart/skchavan@oracle
//
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <utime.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

//#define DEBUG
#define CPU_BENCH
#define NOCUDA

#define MEM_LOGN 28
//#define GATHER2

#define FULLMEM
//#define VERIF

// README
// USE THIS TO CHANGE THE SIZE OF THE STRUCTURE
#define INPUT_SIZE 32  // make sure it's divisible by 8

// max array sizes for certain inputs; going over will cause program to crash
enum {
#if INPUT_SIZE>128  // 512 B, max 19
  rows = 1U << 16,
  array = 1U << 16,
#elif INPUT_SIZE>32 // 128 B, max 21
  rows = 1U << 16,
  array = 1U << 16,
#elif INPUT_SIZE>0  // 32 B, max 23
  rows = 1U << 10,
  array = 1U << 10,
#endif
  groups = 1U << 10,
  segment_bits = 12,
  segments = array / (1U << segment_bits)
};

// each Row stucture is 8 bytes
struct Row {
  unsigned int measure;
  unsigned int group;
};

// stores an array of rows to act as a sized byte container
// i.e. struct Row rows_arr[128/8] is 128 bytes
struct Row16 {
  // [input size/size of Row]
  struct Row rows_arr[INPUT_SIZE/8];
};
	
#ifdef NOCUDA
// ikimasu
//__device__ struct Row d_A[array];
//__device__ unsigned int d_in[rows];
//__device__ struct Row d_out[rows];
//__device__ unsigned long long d_agg1[groups];
//__device__ unsigned long long d_agg2[groups];
//__device__ struct Row d_out2[rows];
//__device__ struct Row * d_B[segments];

__device__ struct Row16 dd_A[array]; // random array
__device__ struct Row16 dd_B[array]; // sequential array
__device__ unsigned int dd_in[rows];
__device__ struct Row16 dd_out[rows];
__device__ struct Row16 dd_out2[rows];

__device__ unsigned long input_size_d = (unsigned long)sizeof(struct Row16); // device input size
__device__ unsigned long row_size_d = (unsigned long)sizeof(struct Row);

unsigned long input_size_h = (unsigned long)sizeof(struct Row16); // host input size

// initialize the GPU arrays
__global__ void d_init()
{
    printf("Initializing data structures.\n");
    int tId = threadIdx.x + (blockIdx.x * blockDim.x);
    hiprandState state;
    hiprand_init((unsigned long long)clock() + tId, 0, 0, &state);
    //printf("Size of word: %lu bytes\n", (unsigned long)sizeof(dd_A[0].str));
    //printf("Size of word container: %lu bytes\n", (unsigned long)sizeof(dd_A[0]));

    // Random fill indirection array A
    unsigned int i;
    unsigned int j;
    printf("Randomly filling array A.\n");
    for (i = 0; i < array; i++) {
      for (j = 0; j < (input_size_d/row_size_d); j++) {
        dd_A[i].rows_arr[j].measure = hiprand_uniform(&state) * array;
        dd_A[i].rows_arr[j].group = hiprand_uniform(&state) * groups;
        //printf("dd_A[%d][%d] - %d\n",i,j,dd_A[i].rows_arr[j].measure);
      }
    }

    printf("Sequentially filling array B.\n");
    for (i = 0; i < array; i++) {
      for (j = 0; j < (input_size_d/row_size_d); j++) {
        dd_B[i].rows_arr[j].measure = array/2;
        dd_B[i].rows_arr[j].group = i & groups;
        //printf("dd_A[%d][%d] - %d\n",i,j,dd_A[i].rows_arr[j].measure);
      }
    }
    //printf("Size of row container: %lu bytes\n", input_size);

    // Random fill input
    printf("Randomly filling input array.\n");
    for (i = 0; i < rows; i++) {
      dd_in[i] = hiprand_uniform(&state) * array;
      //printf("dd_in[%d] - %d\n",i,dd_in[i]);
    }
    printf("Successfully initialized input array.\n");

    // generate random array for benching writes
    //for (i = 0; i < rows; i++) {
    //  dd_out[i] = dd_out2[dd_in[i]];
    //}
}

// bench gathers
__global__ void d_bench()
{
  unsigned i;
  for (i = 0; i < rows; i++) {
    dd_out[i] = dd_A[dd_in[i]];
  }
}

// read / write methods //
// bench random reads
__global__ void d_bench_read_random()
{
  unsigned i;
  struct Row16 temp;
  for (i = 0; i < rows; i++) {
    temp = dd_A[dd_in[i]];
    //dd_out[dd_in[i]] = temp;
  }
}

// bench random writes
__global__ void d_bench_write_random()
{
  unsigned i;
  struct Row16 temp = dd_A[0];
  for (i = 0; i < rows; i++) {
    dd_out2[dd_in[i]] = temp;
    //temp = dd_A[dd_in[i]];
  }
}

// bench linear reads
__global__ void d_bench_read_linear()
{
  unsigned i;
  struct Row16 temp;
  for (i = 0; i < rows; i++) {
    temp = dd_B[i];
    //dd_out[i] = temp;
  }
}

// bench linear writes
__global__ void d_bench_write_linear()
{
  unsigned i;
  struct Row16 temp = dd_A[0];
  for (i = 0; i < rows; i++) {
    dd_out[i] = temp;
    //temp = dd_B[i];
  }
}

#endif // !1

#ifdef VERIF
static __global__ void
d_check(size_t n, benchtype *t)
{
	for (i = 0; i < groups; i++) {
		if (d_agg1[i] != d_agg2[i]) printf("Agg doesn't match: %d\n", i);
	}
}
#endif // VERIF

// helper function for converting from B/ms to MB/s for print output
float convert_to_MBs(float ms) {
  return (input_size_h/1048576.f)/((ms/1000)/rows); // 1048576 = 1024 * 1024, i.e. bytes to MB
}

int main(int argc, char** argv) {
#ifdef NOCUDA
  int ndev;
  hipGetDeviceCount(&ndev);
  int dev = 0;
  //unsigned num_sm = 1; // 1, 2, 4, 8 // # of SMs

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, dev);
  hipSetDevice(dev);

  printf("Using GPU %d of %d GPUs.\n", dev, ndev);
  printf("Warp size = %d.\n", prop.warpSize);
  printf("Multi-processor count = %d.\n", prop.multiProcessorCount);
  printf("Max threads per multi-processor = %d.\n", prop.maxThreadsPerMultiProcessor);
  printf("Grid Size = %d.\n", prop.multiProcessorCount * (prop.maxThreadsPerMultiProcessor / prop.warpSize));
  printf("Thread Size = %d.\n", prop.warpSize);

  dim3 grid(prop.multiProcessorCount * (prop.maxThreadsPerMultiProcessor / prop.warpSize));
  dim3 thread(prop.warpSize);

  printf("Size of word container: %lu bytes\n", input_size_h);
  //printf("Number of SMs: %d\n", num_sm);

  printf("Initializing arrays on GPU with %d elements.\n", array);
  // << <# blocks per grid, # threads per block> >>
  // max = << <65536,1024> >>
  d_init << <8192, 2048>> >();
  //d_bench_write_initialize << <8192, 2048>> >();
  unsigned blocks_per_grid, threads_per_block;
  blocks_per_grid = 1;
  threads_per_block = 1;

  // single threaded
  hipEvent_t read_begin, read_end, write_begin, write_end;
  hipEventCreate(&read_begin);
  hipEventCreate(&read_end);
  hipEventCreate(&write_begin);
  hipEventCreate(&write_end);

  float ms_read_linear, ms_write_linear, ms_read_random, ms_write_random;

  // random read/write //
  printf("Benching random reads.\n");  // random reads
  hipEventRecord(read_begin);
  hipEventSynchronize(read_begin);
  d_bench_read_random << <blocks_per_grid, threads_per_block>> >();
  hipEventRecord(read_end);
  hipEventSynchronize(read_end);

  printf("Benching random writes.\n");  // random writes
  hipEventRecord(write_begin);
  hipEventSynchronize(write_begin);
  d_bench_write_random << <blocks_per_grid, threads_per_block>> >();
  hipEventRecord(write_end);
  hipEventSynchronize(write_end);

  // print random read rate
  hipEventElapsedTime(&ms_read_random, read_begin, read_end);
  printf("%lu-byte random read average = %.6f ms; ", input_size_h, (ms_read_random)/rows);
  printf("rate = %.3f MB/s.\n", convert_to_MBs(ms_read_random));
  
  // print random write rate
  hipEventElapsedTime(&ms_write_random, write_begin, write_end);
  printf("%lu-byte random write average = %.6f ms; ", input_size_h, (ms_write_random)/rows);
  printf("rate = %.3f MB/s.\n", convert_to_MBs(ms_write_random));


  // linear read/write //
  printf("Benching linear reads.\n");  // linear reads
  hipEventRecord(read_begin);
  hipEventSynchronize(read_begin);
  d_bench_read_linear << <blocks_per_grid, threads_per_block>> >();
  hipEventRecord(read_end);
  hipEventSynchronize(read_end);

  printf("Benching linear writes.\n");  // linear writes
  hipEventRecord(write_begin);
  hipEventSynchronize(write_begin);
  d_bench_write_linear << <blocks_per_grid, threads_per_block>> >();
  hipEventRecord(write_end);
  hipEventSynchronize(write_end);

  // print linear read rate
  hipEventElapsedTime(&ms_read_linear, read_begin, read_end);
  printf("%lu-byte linear read average = %.6f ms; ", input_size_h, (ms_read_linear)/rows);
  printf("rate = %.3f MB/s.\n", convert_to_MBs(ms_read_linear));

  // print linear write rate
  hipEventElapsedTime(&ms_write_linear, write_begin, write_end);
  printf("%lu-byte linear write average = %.6f ms; ", input_size_h, (ms_write_linear)/rows);
  printf("rate = %.3f MB/s.\n", convert_to_MBs(ms_write_linear));

  hipEventDestroy(write_end);
  hipEventDestroy(write_begin);
  hipEventDestroy(read_end);
  hipEventDestroy(read_begin);
  printf("Elapsed time = %.6f seconds.\n", (ms_read_linear + ms_write_linear + ms_read_random + ms_write_random)/1000);

  //double time = ms * 1.0e-3;
  //printf("GPU elapsed time = %.6f seconds.\n", time);
  

#endif // !1

#ifdef VERIF
  //d_check << <grid, thread >> >(n, d_t);
  //cpu_bench();
#endif // VERIF

  /**
  printf("Copying host arrays from device.\n");
  checkCudaErrors(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(d_A), sizeof(d_A)));
  //checkCudaErrors(hipMemcpyFromSymbol(h_B, HIP_SYMBOL(d_B), sizeof(d_B)));
  checkCudaErrors(hipMemcpyFromSymbol(h_in, HIP_SYMBOL(d_in), sizeof(d_in)));
  //checkCudaErrors(hipMemcpyFromSymbol(h_out, HIP_SYMBOL(d_out), sizeof(d_out)));
  //checkCudaErrors(hipMemcpyFromSymbol(h_out2, HIP_SYMBOL(d_out2), sizeof(d_out2)));
  //checkCudaErrors(hipMemcpyFromSymbol(h_agg1, HIP_SYMBOL(d_agg1), sizeof(d_agg1)));
  //checkCudaErrors(hipMemcpyFromSymbol(h_agg2, HIP_SYMBOL(d_agg2), sizeof(d_agg2)));
  printf("Successfully copied GPU arrays.\n");**/

#ifdef NOCUDA

  //hipFree(d_A);
  //hipFree(d_in);
  //hipFree(d_out);
  //hipFree(d_out2);
  //hipFree(d_agg1);
  //hipFree(d_agg2);

  hipFree(dd_A);
  hipFree(dd_B);
  hipFree(dd_in);
  hipFree(dd_out);
  hipFree(dd_out2);

#endif // !1
  //unsigned i;
/**
#ifdef CPU_BENCH
  printf("Beginning CPU benchmark.\n");
  struct timeval t0, t1;
  gettimeofday(&t0, 0);
  // Gather rows
  for (i = 0; i < rows; i++) {
          h_out[i] = h_A[h_in[i]];
  }
  // Indirect Gather rows
  for (i = 0; i < rows; i++) {
          h_out[i] = h_A[h_A[h_in[i]].measure];
  }

  // Fused gather group
  for (i = 0; i < rows; i++) {
          h_agg2[h_A[h_in[i]].group] += h_A[h_in[i]].measure;
#ifdef DEBUG
          printf("CPU:  h_agg2[h_A[h_in[i]].group]  = %d\n", h_agg2[h_A[h_in[i]].group]);
#endif // DEBUG
  }
  gettimeofday(&t1, 0);
  printf("CPU bench successful.\n");
  long elapsed = ((t1.tv_sec-t0.tv_sec)*1000000 + t1.tv_usec-t0.tv_usec);
  printf("CPU elapsed time = %lu microseconds.\n", elapsed);

#endif // CPU_BENCH
**/
  return 0;
}
