#include "hip/hip_runtime.h"
//
// Author     :  matto@xilinx 14JAN2018, alai@xilinx 25JULY2018
// Filename   :  indirectTest.cu
// Description:  Cuda random access benchmark example based on indirect.c by gswart/skchavan@oracle
//
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <utime.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define DEBUG
#define CPU_BENCH
#define NOCUDA

#define MEM_LOGN 28
//#define GATHER2

#define FULLMEM
//#define VERIF

#ifdef FULLMEM
enum {
	rows = 1U << 10, // above 18 for rows or arrays causes segfault
        array = 1U << 10,
        rows_test = 1U << 26,
        array_test = 1U << 26,
	groups = 1U << 18,
	segment_bits = 18,
	segments = array / (1U << segment_bits)
};
#else // FULLMEM
enum {
	rows = 1U << 6,
	array = 1U << 6,
	groups = 1U << 4,
	segment_bits = 6,
	segments = array / (1U << segment_bits)
};
#endif
struct Row {
	unsigned int measure;
	unsigned int group;
};


	
#ifdef NOCUDA
// ikimasu
//void *rows2;
//hipError_t error = hipMalloc(&rows2, (1U << 31));
//__device__ struct Row d_A[array];
__device__ struct Row d_A[array_test];
//__device__ struct Row *d_A;
//  hipError_t error = hipMalloc((void**) &d_A, (array_test*sizeof(struct Row)) );
//__device__ unsigned int d_in[rows];
__device__ unsigned int d_in[rows_test];
//__device__ struct Row d_out[rows];
__device__ struct Row d_out[rows_test];
__device__ unsigned long long d_agg1[groups];
__device__ unsigned long long d_agg2[groups];
//__device__ struct Row d_out2[rows];
__device__ struct Row d_out2[rows_test];
//__device__ struct Row * d_B[segments];

__global__ void d_bench()
{
	// ikimasu
	//struct Row A[array];

        //unsigned int in[rows];
        //struct Row out[rows];
        //unsigned long long agg1[groups];
        //unsigned long long agg2[groups];

        //struct Row out2[rows];
        //struct Row * B[segments];

	int tId = threadIdx.x + (blockIdx.x * blockDim.x);
	hiprandState state;
	hiprand_init((unsigned long long)clock() + tId, 0, 0, &state);

  	printf("Initializing data structures.\n");

  	// Random fill indirection array A
  	unsigned int i;
	printf("Random filling A.\n");
  	for (i = 0; i < array_test; i++) {
          //d_A[i].measure = rand() % array_test;
          //d_A[i].group = rand() % groups;
	  d_A[i].measure = hiprand_uniform(&state) * array_test;
          d_A[i].group = hiprand_uniform(&state) * groups;
	  //printf("%d\n",d_A[i].measure);
	  //printf("%d\n",d_A[i].group);

	  //d_A[i].measure = i;
          //d_A[i].group = i % groups;
	  printf("d_A[%d] - %d\n",i,d_A[i].measure);
          //printf("%d\n",d_A[i].group);
  	}

  	// Fill segmented array B
  	/**for (i = 1; i <= segments; i++) {
          d_B[i] = &(d_A[i * (1U << segment_bits)]);
  	}**/

  	// Random fill input
	printf("Random filling input.\n");
  	for (i = 0; i < rows_test; i++) {
          //d_in[i] = rand() % array_test;
	  d_in[i] = hiprand_uniform(&state) * rows_test;
	  //printf("%d\n",d_in[i]);

	  //d_in[i] = i;
	  printf("d_in[%d] - %d\n",i,d_in[i]);
	}

  	// Zero aggregates
  	for (i = 0; i < groups; i++) {
          d_agg1[i] = 0;
          d_agg2[i] = 0;
  	}
	//unsigned int i;

	// Gather rows
	for (i = 0; i < rows; i++) {
		d_out[i] = d_A[d_in[i]];
	}

	// Indirect Gather rows
	for (i = 0; i < rows; i++) {
		d_out[i] = d_A[d_A[d_in[i]].measure]; 
	}

	// Fused gather group
	for (i = 0; i < rows; i++) {
		d_agg2[d_A[d_in[i]].group] += d_A[d_in[i]].measure;
#ifdef DEBUG
		printf("GPU: d_agg2[d_A[d_in[i]].group]  = %d\n", d_agg2[d_A[d_in[i]].group] );
#endif // DEBUG
	}

#ifdef GATHER2
	// Segmented gather
	for (i = 0; i < rows; i++) {
		int segment_number = (d_in[i] >> segment_bits);
		int segment_offset = (d_in[i] & ((1U << segment_bits) - 1));
#ifdef DEBUG
		printf("d_in[i] = %d\n", d_in[i]);
		printf("segment_number = %d\n", segment_number);
		printf("segment_offset = %d\n", segment_offset);
		printf("d_B[0] = %d\n", d_B[0]);
		printf("d_B[segment_number][segment_offset] = %d\n", d_B[segment_number][segment_offset]);
		printf("d_out2[i] = %d\n", d_out2[i]);
#endif // DEBUG

		d_out2[i] = d_B[segment_number][segment_offset];
	}
#endif // GATHER2


}
#endif // !1

#ifdef VERIF
static __global__ void
d_check(size_t n, benchtype *t)
{
	for (i = 0; i < groups; i++) {
		if (d_agg1[i] != d_agg2[i]) printf("Agg doesn't match: %d\n", i);
	}
}
#endif // VERIF


static void
init()
{
	struct Row A[array];

	unsigned int in[rows];
	struct Row out[rows];
	unsigned long long agg1[groups];
	unsigned long long agg2[groups];

	struct Row out2[rows];
	struct Row * B[segments];

  printf("Initializing data structures.\n");

  // Random fill indirection array A
  unsigned int i;
  for (i = 0; i < array; i++) {
	  A[i].measure = rand() % array;
	  A[i].group = rand() % groups;
  }

  // Fill segmented array B
  for (i = 0; i < segments; i++) {
	  B[i] = &(A[i * (1U << segment_bits)]);
  }

  // Random fill input
  for (i = 0; i < rows; i++)
	  in[i] = rand() % array;

  // Zero aggregates
  for (i = 0; i < groups; i++) {
	  agg1[i] = 0;
	  agg2[i] = 0;
  }

#ifdef NOCUDA
  // ikimasu
  //struct Row *Acpy = new struct Row[array];
  //std::copy(A, A+array, Acpy);
  //for (i = 0; i < array; i++) {
  //  printf("woohoo");
  //  Acpy[i] = A[i];
    //d_A[i] = A[i];
    //hipMemcpyToSymbol(HIP_SYMBOL(d_A[i]), A[i], (array*sizeof(A[i])));
  //}
  //checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_A), &Acpy, sizeof(Acpy)));

  //checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_A), A, sizeof(A)));
  //checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_B), B, sizeof(B)));
  //checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_in), in, sizeof(in)));
  //checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_out), out, sizeof(out)));
  //checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_out2), out2, sizeof(out2)));
  //checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_agg1), agg1, sizeof(agg1)));
  //checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_agg2), agg2, sizeof(agg2)));
#endif // !1


#ifdef CPU_BENCH

  // Gather rows
  for (i = 0; i < rows; i++) {
	  out[i] = A[in[i]];
  }

  // Indirect Gather rows
  for (i = 0; i < rows; i++) {
	  out[i] = A[A[in[i]].measure];
  }

  // Fused gather group
  for (i = 0; i < rows; i++) {
	  agg2[A[in[i]].group] += A[in[i]].measure;
#ifdef DEBUG
	  printf("CPU:  agg2[A[in[i]].group]  = %d\n", agg2[A[in[i]].group]);
#endif // DEBUG  
#endif // CPU_BENCH
  }

}

int
main(int argc, char *argv[])
{
  
  init();

#ifdef NOCUDA
  int ndev;
  hipGetDeviceCount(&ndev);
  int dev = 0;

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, dev);
  hipSetDevice(dev);

  printf("Using GPU %d of %d GPUs.\n", dev, ndev);
  printf("Warp size = %d.\n", prop.warpSize);
  printf("Multi-processor count = %d.\n", prop.multiProcessorCount);
  printf("Max threads per multi-processor = %d.\n", prop.maxThreadsPerMultiProcessor);
  printf("Grid Size = %d.\n", prop.multiProcessorCount * (prop.maxThreadsPerMultiProcessor / prop.warpSize));
  printf("Thread Size = %d.\n", prop.warpSize);

  dim3 grid(prop.multiProcessorCount * (prop.maxThreadsPerMultiProcessor / prop.warpSize));
  dim3 thread(prop.warpSize);

  hipEvent_t begin, end;
  hipEventCreate(&begin);
  hipEventCreate(&end);

  hipEventRecord(begin);
  hipEventSynchronize(begin);

  //d_bench << <grid, thread >> >();
  d_bench << <1, 1 >> >();

  hipEventRecord(end);
  hipEventSynchronize(end);

  float ms;
  hipEventElapsedTime(&ms, begin, end);
  hipEventDestroy(end);
  hipEventDestroy(begin);
  double time = ms * 1.0e-3;
  printf("Elapsed time = %.6f seconds.\n", time);

#endif // !1

#ifdef VERIF
  //d_check << <grid, thread >> >(n, d_t);
  cpu_bench();
#endif // VERIF

#ifdef NOCUDA
  //hipFree(rows2);

  hipFree(d_A);
  //hipFree(d_B);
  hipFree(d_in);
  hipFree(d_out);
  hipFree(d_out2);
  hipFree(d_agg1);
  hipFree(d_agg2);

#endif // !1

  return 0;
}
